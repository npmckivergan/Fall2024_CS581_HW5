/*  
Name:   Nolan McKivergan
Email:  npmckivergan@crimson.ua.edu
Course: CS 581
Homework #: 5
Instructions to compile the program: nvcc gpu_paper.cu -O3 -o gpu_paper
Instructions to run the program: ./gpu_paper <grid size> <max gens> <output file path>
*/ 

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <string.h>

#define TILE_DIM 16

int size;
int max_gens;
char **curr_matrix;
char **next_matrix;

void print_board(char **matrix, int size, FILE *output_file);
double gettime();

// Kernel function
__global__ void updateKernel(char *d_curr_matrix, char *d_next_matrix, int width, int height, int *d_change_flag) {
    __shared__ char tile[TILE_DIM + 2][TILE_DIM + 2];
    __shared__ int local_change_flag;

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int tx = threadIdx.x + 1;
    int ty = threadIdx.y + 1;

    // Initialize local change flag
    if (threadIdx.x == 0 && threadIdx.y == 0) local_change_flag = 0;
    __syncthreads();

    // Load cell and neighbors into shared memory
    if (x < width && y < height) {
        tile[ty][tx] = d_curr_matrix[y * width + x];
        if (threadIdx.x == 0) tile[ty][0] = (x > 0) ? d_curr_matrix[y * width + (x - 1)] : 0;
        if (threadIdx.x == TILE_DIM - 1) tile[ty][TILE_DIM + 1] = (x < width - 1) ? d_curr_matrix[y * width + (x + 1)] : 0;
        if (threadIdx.y == 0) tile[0][tx] = (y > 0) ? d_curr_matrix[(y - 1) * width + x] : 0;
        if (threadIdx.y == TILE_DIM - 1) tile[TILE_DIM + 1][tx] = (y < height - 1) ? d_curr_matrix[(y + 1) * width + x] : 0;
        if (threadIdx.x == 0 && threadIdx.y == 0) tile[0][0] = (x > 0 && y > 0) ? d_curr_matrix[(y - 1) * width + (x - 1)] : 0;
        if (threadIdx.x == TILE_DIM - 1 && threadIdx.y == 0) tile[0][TILE_DIM + 1] = (x < width - 1 && y > 0) ? d_curr_matrix[(y - 1) * width + (x + 1)] : 0;
        if (threadIdx.x == 0 && threadIdx.y == TILE_DIM - 1) tile[TILE_DIM + 1][0] = (x > 0 && y < height - 1) ? d_curr_matrix[(y + 1) * width + (x - 1)] : 0;
        if (threadIdx.x == TILE_DIM - 1 && threadIdx.y == TILE_DIM - 1) tile[TILE_DIM + 1][TILE_DIM + 1] = (x < width - 1 && y < height - 1) ? d_curr_matrix[(y + 1) * width + (x + 1)] : 0;
    }

    __syncthreads();

    // Process cell
    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int numNeighbors = tile[ty - 1][tx - 1] + tile[ty - 1][tx] + tile[ty - 1][tx + 1] +
                           tile[ty][tx - 1] + tile[ty][tx + 1] +
                           tile[ty + 1][tx - 1] + tile[ty + 1][tx] + tile[ty + 1][tx + 1];

        int idx = y * width + x;
        if (tile[ty][tx] == 1) {
            if (numNeighbors < 2 || numNeighbors > 3) {
                d_next_matrix[idx] = 0;
                local_change_flag = 1;
            } else {
                d_next_matrix[idx] = 1;
            }
        } else {
            if (numNeighbors == 3) {
                d_next_matrix[idx] = 1;
                local_change_flag = 1;
            } else {
                d_next_matrix[idx] = 0;
            }
        }
    }

    __syncthreads();

    // Update global change flag
    if (threadIdx.x == 0 && threadIdx.y == 0 && local_change_flag) {
        atomicExch(d_change_flag, 1);
    }
}

// Game of Life function
void gameOfLife(char *grid, int width, int height, int steps) {
    char *d_curr_matrix, *d_next_matrix;
    int *d_change_flag;
    size_t size = width * height * sizeof(char);

    hipMalloc(&d_curr_matrix, size);
    hipMalloc(&d_next_matrix, size);
    hipMalloc(&d_change_flag, sizeof(int));
    hipMemcpy(d_curr_matrix, grid, size, hipMemcpyHostToDevice);

    dim3 blockSize(TILE_DIM, TILE_DIM);
    dim3 gridSize((width + TILE_DIM - 1) / TILE_DIM, (height + TILE_DIM - 1) / TILE_DIM);

    for (int step = 0; step < steps; ++step) {
        int change_flag = 0;
        hipMemcpy(d_change_flag, &change_flag, sizeof(int), hipMemcpyHostToDevice);

        updateKernel<<<gridSize, blockSize>>>(d_curr_matrix, d_next_matrix, width, height, d_change_flag);

        hipMemcpy(&change_flag, d_change_flag, sizeof(int), hipMemcpyDeviceToHost);
        if (change_flag == 0) break;

        std::swap(d_curr_matrix, d_next_matrix);
    }

    hipMemcpy(grid, d_curr_matrix, size, hipMemcpyDeviceToHost);
    hipFree(d_curr_matrix);
    hipFree(d_next_matrix);
    hipFree(d_change_flag);
}

int main(int argc, char **argv) {
    if (argc < 3) {
        printf("Usage: ./a.exe <matrix size> <max generations> <output file path>\n");
        return 1;
    }
    size = atoi(argv[1]) + 2;
    max_gens = atoi(argv[2]);
    const char *output_file = argv[3];
    FILE *file = fopen(output_file, "w");

    curr_matrix = (char**)malloc(size * sizeof(char*));
    next_matrix = (char**)malloc(size * sizeof(char*));
    for (int x = 0; x < size; x++) {
        curr_matrix[x] = (char*)malloc(size * sizeof(char));
        next_matrix[x] = (char*)malloc(size * sizeof(char));
    }

    for (int x = 0; x < size; x++) {
        for (int y = 0; y < size; y++) {
            curr_matrix[x][y] = 0;
        }
    }

    for (int x = 1; x < size - 1; x++) {
        for (int y = 1; y < size - 1; y++) {
            curr_matrix[x][y] = rand() % 2;
        }
    }

    char *grid = new char[size * size];
    for (int x = 0; x < size; x++) {
        for (int y = 0; y < size; y++) {
            grid[x * size + y] = curr_matrix[x][y];
        }
    }

    double start_time = gettime();
    gameOfLife(grid, size, size, max_gens);
    double end_time = gettime();
    printf("Time taken: %lf seconds\n", end_time - start_time);

    for (int x = 0; x < size; x++) {
        for (int y = 0; y < size; y++) {
            curr_matrix[x][y] = grid[x * size + y];
        }
    }

    print_board(curr_matrix, size, file);

    for (int x = 0; x < size; x++) {
        free(curr_matrix[x]);
        free(next_matrix[x]);
    }
    free(curr_matrix);
    free(next_matrix);
    delete[] grid;
    return 0;
}

void print_board(char **matrix, int size, FILE *output_file) {
        for (int x = 1; x < size - 1; x++) {
            for (int y = 1; y < size - 1; y++) {
                fprintf(output_file, "%c ", matrix[x][y] ? 'O' : '.');
            }
            fprintf(output_file, "\n");
        }
        fprintf(output_file, "\n");
}

double gettime(void) {
    struct timeval tval;
    gettimeofday(&tval, NULL);
    return (double)tval.tv_sec + (double)tval.tv_usec / 1000000.0;
}
